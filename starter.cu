//@@ The purpose of this code is to become familiar with the submission
//@@ process. Do not worry if you do not understand all the details of
//@@ the code.

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
  int deviceCount;

  hipGetDeviceCount(&deviceCount);
  printf("Device Count %d\n", deviceCount);

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        printf("No CUDA GPU has been detected\n");
        return -1;
      } else if (deviceCount == 1) {
        //@@ WbLog is a provided logging API (similar to Log4J).
        //@@ The logging function wbLog takes a level which is either
        //@@ OFF, FATAL, ERROR, WARN, INFO, DEBUG, or TRACE and a
        //@@ message to be printed.
        printf("There is 1 device supporting CUDA\n");
      } else {
        printf("There are %d devices supporting CUDA\n", deviceCount);
      }
    }

    printf("Device %d  name: %s\n", dev, deviceProp.name);
   // wbLog(TRACE, " Computational Capabilities: ", deviceProp.major, ".",
   //       deviceProp.minor);
    printf(" Maximum global memory size: %lu\n",deviceProp.totalGlobalMem);
    printf(" Maximum constant memory size: %lu\n", deviceProp.totalConstMem);
    printf(" Maximum shared memory size per block: %lu\n", deviceProp.sharedMemPerBlock);
    printf(" Maximum block dimensions: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf(" Maximum grid dimensions: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf(" Warp size: %d\n", deviceProp.warpSize);
  }

  return 0;
}
